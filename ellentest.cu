#include "hip/hip_runtime.h"
// Ellen test
//compile when makefile is being weird: 
///usr/local/cuda/bin/nvcc -I/usr/local/cuda/include -L /usr/local/cuda/lib64 -lcuda -lcudart -lm -o EllenTest ellentest.cu

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
//#include <complex.h> //for complex numbers

//testing xy 4*2
__global__ void hermitian_transpose(const float2* input_h, float2* output_hh, int N, int K) { //const because we do not want to modify the input matrix!!!
	int col = threadIdx.x + blockDim.x * blockIdx.x; //find what col and row this thread is responsible for
	int row = threadIdx.y + blockDim.y * blockIdx.y;	//ex 0,0 or 1,3

    if (col < N && row < K) {
		//translate from ex 1,3 to index 1+3*2 = 7
        int idx_in = col + row * N; //what index we are on in matrix
		//1,3 to instead 3,1 : index 3+1*4=7
        int idx_out = row + col * K; //output should be reversed (transpose)

	//conjugate here - in a float2: .x is the real part, .y is imaginary part
        output_hh[idx_out].x = input_h[idx_in].x; //conjugate
        output_hh[idx_out].y = -input_h[idx_in].y; //conjugate, it is negative for the imaginary part
    }
}

//A size (K, M)
//B size (K, N)
//C size ((N or K), M)
//B*A = C dont know why is flipped
/*__global__ void complex_matrix_mult(const float2* A, const float2* B, float2* C, const int M, const int K, const int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float2 sum = make_float2(0.0f, 0.0f);

        for (int k = 0; k < K; k++) {
            float2 a = A[row * K + k];
            float2 b = B[k * N + col];

            float real_part = a.x * b.x - a.y * b.y;
            float imag_part = a.x * b.y + a.y * b.x;

            sum.x += real_part;
            sum.y += imag_part;
        }

        C[row * N + col] = sum;
    }
}*/
//axb * cxd = axd
//b=c otherwise matmul cant happen
//K*M * N*1 = K*1
__global__ void complex_matrix_mult(const float2* A, const float2* B, float2* C, const int res_row, const int a_row_b_col, const int res_col) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < res_row && col < res_col) {
        float2 sum = make_float2(0.0f, 0.0f);

        for (int k = 0; k < a_row_b_col; k++) {
            float2 a = A[row * a_row_b_col + k]; //
            float2 b = B[k * res_col + col];

            float real_part = a.x * b.x - a.y * b.y;
            float imag_part = a.x * b.y + a.y * b.x;

            sum.x += real_part;
            sum.y += imag_part;
        }

        C[row * res_col + col] = sum;
		//if column done (col == K)- set event for cholesky?
    }
}

int main(int argc, char *argv[])  {

/*
* ellen test
*/

	int Grid_Dim_x=1, Grid_Dim_y=1;			//Grid structure values
	int Block_Dim_x=1, Block_Dim_y=1;		//Block structure values

	int noThreads_x, noThreads_y;		// number of threads available in device, each dimension
	int noThreads_block;				// number of threads in a block

	int N = 4;  		//antennas
	int K = 2;			//users

	float2 h[N*K] = { {1.0f, 2.0f}, {3.0f, 4.0f}, {5.0f, 6.0f}, {7.0f, 8.0f}, {9.0f, 10.0f}, {11.0f, 12.0f}, {13.0f, 14.0f}, {15.0f, 16.0f} };
	float2 y[N] =  { {1.0f, 3.0f}, {4.0f, 8.0f}, {16.0f, 8.0f}, {2.0f, 2.0f} };//2x1 vector
	//is y complex or not?
	
	for (int i = 0; i < N*K; i++) { //print input matrix
		printf("(%f + %fi)\n", h[i].x, h[i].y);
	}
	
	float2 *mat_h,*mat_hh,*mat_hhh, *vec_y, *vec_hy; //float2 z = {1.0f, 2.0f}; // z = 1.0 + 2.0i
	int mat1_size;					// number of bytes in arrays
	int mat2_size;
	int vec1_size;
	int vec2_size;
	
	hipEvent_t start, stop;     		// using cuda events to measure time
	float elapsed_time_ms;       		// which is applicable for asynchronous code also

/* --------------------ENTER INPUT PARAMETERS AND DATA -----------------------*/
		
	Grid_Dim_x = 1;
	Grid_Dim_y = 1;
	Block_Dim_x = 4;
	Block_Dim_y = 2;

	noThreads_x = Grid_Dim_x * Block_Dim_x;		// number of threads in x dimension
	noThreads_y = Grid_Dim_y * Block_Dim_y;		// number of threads in y dimension

	noThreads_block = Block_Dim_x * Block_Dim_y;	// number of threads in a block

	dim3 Grid(Grid_Dim_x, Grid_Dim_x);		//Grid structure
	dim3 Block(Block_Dim_x,Block_Dim_y);	//Block structure, threads/block limited by specific device

	mat1_size = K * N * sizeof(float2); //Hh and H are K*N and N*K
	mat2_size = K * K * sizeof(float2);	//gramian is K*K
	vec1_size = N * sizeof(float2);		//vec is K*1
	vec2_size = K * sizeof(float2);

/* ------------- COMPUTATION DONE ON GPU ----------------------------*/

	hipMalloc((void**)&mat_h, mat1_size);		// allocate memory on device
	hipMalloc((void**)&mat_hh, mat1_size);
	hipMalloc((void**)&mat_hhh, mat2_size);
	hipMalloc((void**)&vec_y, vec1_size); //size of y vector is Nx1
	hipMalloc((void**)&vec_hy, vec2_size); //size of Hy vector is Nx1

	hipMemcpy(mat_h, h, mat1_size ,hipMemcpyHostToDevice); //put h in device
	hipMemcpy(vec_y, y, vec1_size ,hipMemcpyHostToDevice); //put y in device

//--------------------------TRANSPOSE-Hh---------------------------------
	hipEventCreate(&start);     		// instrument code to measure start time
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	hermitian_transpose<<<Grid,Block>>>(mat_h,mat_hh,N,K); //calc hermitian Hh

	float2 output[N*K];//just to print, device has mat_hh, host does not need it?
	hipMemcpy(output, mat_hh, mat1_size, hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);     	// instrument code to measure end time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );

	for (int i = 0; i < N*K; i++) {
		printf("(%f + %fi)\n", output[i].x, output[i].y);
	}
	
	printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms);  // print out execution time

//-------------------------MATMUL-HhH--------------------------------------
	//a is H, output is Hh
	//this is HhH
	//complex_matrix_mult<<<Grid,Block>>>(mat_hh, mat_h, mat_hhh, K,N,K); //why no work
	complex_matrix_mult<<<Grid,Block>>>(mat_h, mat_hh, mat_hhh, K,N,K);//A, B, C, res_row, a_row_b_col, res_col

	float2 gramian[K*K];
	hipMemcpy(gramian, mat_hhh, mat2_size, hipMemcpyDeviceToHost);

	printf("gramian\n");
	for (int i = 0; i < K*K; i++) {
		printf("(%f + %fi)\n", gramian[i].x, gramian[i].y);
	}
	
//-------------------------MAT-VEC-MUL-Hy--------------------------------------
	//this is Hhy
	//complex_matrix_mult<<<Grid,Block>>>(mat_hh, vec_y, vec_hy, K,N,1); //why does this not work???
	complex_matrix_mult<<<Grid,Block>>>(vec_y, mat_hh, vec_hy, 1,N,K);//WHY IS IT FLIPPED
	
	float2 hy[N];
	hipMemcpy(hy, vec_hy, vec2_size, hipMemcpyDeviceToHost);
	
	printf("Hy\n");
	for (int i = 0; i < K; i++) {
		printf("(%f + %fi)\n", hy[i].x, hy[i].y);
	}

/* --------------  clean up  ---------------------------------------*/
	hipFree(mat_h);
	hipFree(mat_hh);
	hipFree(mat_hhh);
	hipFree(vec_y);
	hipFree(vec_hy);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
