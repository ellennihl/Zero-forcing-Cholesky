
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_complex.h>

/**
	This metod takes in a matrix and returns the hermetian transpose of the matrix	
	input_h is the input matrix with size NxK
	output_hh is the resulting matrix with size KxN
	N is the nr of in input_h
	K the nr of rows in input_h
*/
__global__ void hermitian_transpose(const float2* input_h, float2* output_hh, int N, int K) { //const because we do not want to modify the input matrix!!!
	
	int row = threadIdx.x + blockDim.x * blockIdx.x; //find what col and row this thread is responsible for
	int col = threadIdx.y + blockDim.y * blockIdx.y;	//ex 0,0 or 1,3
	
	if (col < K && row < N) {
		int idx_in = col * N + row;
		int idx_out = row * K + col;
		//printf("(%d,%d)  in: %d, out: %d\n",row,col,idx_in,idx_out);
		//conjugate here - in a float2: .x is the real part, .y is imaginary part
        output_hh[idx_out].x = input_h[idx_in].x; //conjugate
        output_hh[idx_out].y = -input_h[idx_in].y; //conjugate, it is negative for the imaginary part
    }
}

/**
	Pree Condition: Same size at Arow/Bcol 
	This funktion calculates the dot produkt of two complex matrixes where A.B=C
	A is the first input matrix
	B is the second input matrix
	C is the resulting matrix
	res_row is the nr of rows in matrix A
	a_col_b_row is the nr of columns of A matrix and nr of rows in B matrix
	res_col is nr of columns in B matrix
*/
__global__ void complex_matrix_mult(const float2* A, const float2* B, float2* C, const int res_row, const int a_col_b_row, const int res_col) {

	int row = threadIdx.x + blockDim.x * blockIdx.x; //find what col and row this thread is responsible for
	int col = threadIdx.y + blockDim.y * blockIdx.y;

	if (row < res_row && col < res_col) {
        float2 sum = make_float2(0.0f, 0.0f);

        for (int k = 0; k < a_col_b_row; k++) {
			//printf("(%d,%d) a: %d   b: %d\n",row,col,row * a_col_b_row + k, k * res_col + col);

            float2 a = A[k * res_row + row]; //column-major!!!!!!
            float2 b = B[col * a_col_b_row + k];
            float real_part = a.x * b.x - a.y * b.y;
            float imag_part = a.x * b.y + a.y * b.x;
            sum.x += real_part;
            sum.y += imag_part;
        }
		C[col * res_row + row] = sum;
	}
}

int main() {

	//Size of matrix N=antennas, K=Users
	//
	int N = 3;
	int K = 3;
	
	hipFloatComplex y[N];
	//initializing y matrix
	y[0].x = -1.15044381816198;
	y[0].y = 2.80297100338098;
	y[1].x = -1.45737148064847;
	y[1].y = 0.105134117295914;
	y[2].x = -2.73160735027786;
	y[2].y = -0.0430050084558768;
	
	//The h stands for host
	hipFloatComplex H[N*K],hHH[K*K], hmHH[K*K];
	//initializing H matrix
	H[0].x = -0.14871528137562;
	H[0].y = -0.839585070157793;
	H[1].x = 0.456796194001739;
	H[1].y = -1.39648740223667;
	H[2].x = -0.627350895700304;
	H[2].y = -0.491338636279611;
	H[3].x = 0.756444232794338;
	H[3].y = -0.238637048003854;
	H[4].x = -0.374235630126775;
	H[4].y = 0.686050058020553;
	H[5].x = 0.959923600007699;
	H[5].y = -0.0923017429928966;
	H[6].x = 1.25391777895517;
	H[6].y = 0.0860634779712874;
	H[7].x = -0.322123665443045;
	H[7].y = -0.101934261054657;
	H[8].x = -0.727806592386333;
	H[8].y = 0.0283459633648643;
	
	//The d stands for device
    hipFloatComplex *dH, *dHH, *dmHH;
    hipMalloc((void **)&dH, N*K*sizeof(hipFloatComplex));
    hipMalloc((void **)&dHH, K*K*sizeof(hipFloatComplex));
	hipMalloc((void **)&dmHH, K*K*sizeof(hipFloatComplex));

    //Copy input data to array on GPU.
    hipMemcpy(dH, H, N*K*sizeof(hipFloatComplex), hipMemcpyHostToDevice);

	//Run the transpose on gpu
	dim3 blockDimsT(N,K);
	dim3 GridDimsT(1);
    hermitian_transpose<<<blockDimsT,GridDimsT>>>(dH, dHH,N,K);
	//Run the multiplication on the GPU
	dim3 blockDims(K,K);
	dim3 GridDims(1);
	complex_matrix_mult<<<blockDims,GridDims>>>(dHH, dH, dmHH,N,K,N);

	//Coppy the results to the host
    hipMemcpy(hHH, dHH, N*K*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    hipMemcpy(hmHH, dmHH, K*K*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);


	//Test for hermetian transpose
	for (int i = 0; i<K; ++i) {
		for (int j = 0; j<K; ++j) {
			printf("%d", H[i*K+j].x == hHH[j*K+i].x);
			printf("%d ", H[i*K+j].y == -hHH[j*K+i].y);
		}
        printf("\n");
    }
	printf("\n");
	
	//Print out the gramian matrix
	for (int i = 0; i<K; ++i) {
		for (int j = 0; j<K; ++j) {
			printf("%f+%fi ", hmHH[i*K+j].x,hmHH[i*K+j].y);
		}
        printf(";\n");
    }
	
    // Free up the arrays on the GPU.
    hipFree(dH);
    hipFree(dHH);
	hipFree(dmHH);
    return 0;
}