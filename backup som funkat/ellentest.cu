#include "hip/hip_runtime.h"
// Ellen test

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
//#include <complex.h> //for complex numbers

__global__ void gpu_matrixadd(int *a,int *b, int *c, int N) {

	int col = threadIdx.x + blockDim.x * blockIdx.x; 
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	int index = row * N + col;

      	if(col < N && row < N)
          c[index] = a[index]+b[index];

}

//testing
__global__ void hermitian_transpose_kernel(const float2* input_h, float2* output_hh, int N) { //const because we do not want to modify the input matrix!!!
	int col = threadIdx.x + blockDim.x * blockIdx.x; //find what col and row this thread is responsible for
	int row = threadIdx.y + blockDim.y * blockIdx.y;

    if (col < N && row < N) {
        int idx_in = col + row * N; //what index we are on in matrix
        int idx_out = row + col * N; //output should be reversed (transpose)

	//conjugate here - in a float2: .x is the real part, .y is imaginary part
        output_hh[idx_out].x = input_h[idx_in].x; //conjugate
        output_hh[idx_out].y = -input_h[idx_in].y; //conjugate, it is negative for the imaginary part
    }
}

__global__ void gpu_matrixmult(int *gpu_a, int *gpu_b, int *gpu_c, int N) {

	int k, sum = 0;
	int col = threadIdx.x + blockDim.x * blockIdx.x; 
	int row = threadIdx.y + blockDim.y * blockIdx.y;

       if(col < N && row < N) {
		for(k = 0; k < N; k++) 
          		sum += gpu_a[row * N + k] * gpu_b[k * N + col];
		gpu_c[row * N + col] = sum;
	}
}

void cpu_matrixadd(int *a,int *b, int *c, int N) {

	int index;
	for(int col=0;col < N; col++) 
		for(int row=0;row < N; row++) {
			index = row * N + col;
           		c[index] = a[index]+b[index];
		}
}

int main(int argc, char *argv[])  {

/*
* ellen test
*/
	int Grid_Dim_x=1, Grid_Dim_y=1;			//Grid structure values
	int Block_Dim_x=1, Block_Dim_y=1;		//Block structure values

	int noThreads_x, noThreads_y;		// number of threads available in device, each dimension
	int noThreads_block;				// number of threads in a block

	int N = 2;  					// size of array in each dimension
	//float2 *a,*b,*c,*d;
	float2 a[N*N] = { {1.0f, 2.0f}, {3.0f, 4.0f}, {5.0f, 6.0f}, {7.0f, 8.0f} };
	
	for (int i = 0; i < N*N; i++) {
		printf("(%f + %fi)\n", a[i].x, a[i].y);
	}
	//float complex *mat_h,*mat_hh,*mat_hhh;
	float2 *mat_h,*mat_hh,*mat_hhh;
	//float2 z = {1.0f, 2.0f}; // z = 1.0 + 2.0i
	int size;					// number of bytes in arrays

	hipEvent_t start, stop;     		// using cuda events to measure time
	float elapsed_time_ms;       		// which is applicable for asynchronous code also

/* --------------------ENTER INPUT PARAMETERS AND DATA -----------------------*/
		
	Grid_Dim_x = 1;
	Grid_Dim_y = 1;
	Block_Dim_x = 2;
	Block_Dim_y = 2;

	noThreads_x = Grid_Dim_x * Block_Dim_x;		// number of threads in x dimension
	noThreads_y = Grid_Dim_y * Block_Dim_y;		// number of threads in y dimension

	noThreads_block = Block_Dim_x * Block_Dim_y;	// number of threads in a block

	dim3 Grid(Grid_Dim_x, Grid_Dim_x);		//Grid structure
	dim3 Block(Block_Dim_x,Block_Dim_y);	//Block structure, threads/block limited by specific device

	size = N * N * sizeof(float2);		// number of bytes in total in arrays

	//a = (float2*) malloc(size);		//this time use dynamically allocated memory for arrays on host
	//b = (float2*) malloc(size);
	//c = (float2*) malloc(size);		// results from GPU
	//d = (float2*) malloc(size);		// results from CPU
	

/*	for(i=0;i < N;i++)			// load arrays with some numbers
	for(j=0;j < N;j++) {
		a[i * N + j] = i;
		b[i * N + j] = i;
	}*/
	//float2 a[N] = { {1.0f, 2.0f}, {3.0f, 4.0f}, {5.0f, 6.0f}, {7.0f, 8.0f} };

/* ------------- COMPUTATION DONE ON GPU ----------------------------*/

	hipMalloc((void**)&mat_h, size);		// allocate memory on device
	hipMalloc((void**)&mat_hh, size);
	hipMalloc((void**)&mat_hhh, size);

	hipMemcpy(mat_h, a , size ,hipMemcpyHostToDevice);
	//hipMemcpy(mat_hh, b , size ,hipMemcpyHostToDevice);
	//hipMemcpy(mat_hhh, c , size ,hipMemcpyHostToDevice);

	hipEventCreate(&start);     		// instrument code to measure start time
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
//	hipEventSynchronize(start);  	// Needed?

	//gpu_matrixmult<<<Grid,Block>>>(mat_h,mat_hh,mat_hhh,N);
	hermitian_transpose_kernel<<<Grid,Block>>>(mat_h,mat_hh,N);

	float2 output[N*N];
	hipMemcpy(output, mat_hh, size, hipMemcpyDeviceToHost);

	//hipMemcpy(c,mat_hhh, size ,hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);     	// instrument code to measue end time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );

	for (int i = 0; i < N*N; i++) {
		printf("(%f + %fi)\n", output[i].x, output[i].y);
	}
	
	printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms);  // print out execution time

/* ------------- COMPUTATION DONE ON HOST CPU ----------------------------*/
/*
	hipEventRecord(start, 0);		// use same timing
//	hipEventSynchronize(start);  	// Needed?

	cpu_matrixadd(a,b,d,N);		// do calculation on host

	hipEventRecord(stop, 0);     	// instrument code to measue end time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );

	printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms);  // print out execution time

/* ------------------- check device creates correct results -----------------*/
/*
	for(i=0;i < N*N;i++) {
		if (c[i] != d[i]) printf("*********** ERROR in results, CPU and GPU create different answers ********\n");
		break;
	}

	printf("\nEnter c to repeat, return to terminate\n");
	scanf("%c",&key);
	scanf("%c",&key);
*/

/* --------------  clean up  ---------------------------------------*/
//	free(a);
//	free(b);
//	free(c);
//	free(d);
	hipFree(mat_h);
	hipFree(mat_hh);
	hipFree(mat_hhh);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
