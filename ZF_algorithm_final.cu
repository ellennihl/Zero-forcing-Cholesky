
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_complex.h>
//#include <omp.h>

/**
	This is to use qsort
*/
int cmpfunc (const void * a, const void * b) {
   return ( *(int*)a - *(int*)b );
}

/**
	Takes a csv containing a matrix and returns an array column major
*/
float2 *read_matrix_from_csv(char filename[], int num_rows, int num_cols) {
	// Allocate memory for the matrix
	float2 *matrix = (float2 *) malloc(num_rows * num_cols * sizeof(float2));
	bool real = true;

	char tempchar[20];
	strcpy(tempchar, filename);
	for(int i=0;i<2;i++){
		if(i == 0){
			strcat(filename, "_real.csv");
			real = true;
		}
		else{
			strcat(tempchar, "_imag.csv");
			strcpy(filename, tempchar);
			real = false;
		}
		
	  FILE *file = fopen(filename, "r");
	  if (file == NULL) {
		fprintf(stderr, "Error: Could not open file '%s'\n", filename);
		exit(1);
	  }
	  // Read the data from the file into the matrix
	  int size = (num_rows*20+1)*sizeof(char);//max 20 chars and "," for one value + "\0" or "\n"
	  char line[size];
	  int row = 0, col = 0;
	  while (fgets(line, size, file) && row < num_rows) {
		if (line[strlen(line) - 1] == '\n') {
		  line[strlen(line) - 1] = '\0';  // Remove newline character
		}

		char *token = strtok(line, ",");
		while (token != NULL && col < num_cols) {
			if(real){
				matrix[row + col * num_rows].x = atof(token); // Change the ordering of the matrix
			}
			else{
				matrix[row + col * num_rows].y = atof(token); // Change the ordering of the matrix
			}
			col++;
			token = strtok(NULL, ",");
		}
		col = 0;
		row++;
	  }
	  fclose(file);
	}
  
  return matrix;
}

/**
	extra calculates how many elements of a matrix each thread needs to calculate of there are too few threads
	elements is the number of elements there is in a row/column in the matrix.
	nrOfThreads are the number of threads available for use
*/
__inline__ __device__ int extra(int elements, int nrOfThreads){
	int tmp = ceil((float)elements/(float)nrOfThreads);
	return tmp;
}

/**
	cuCsqrt takes in a complex number and returns the square root of this number
	z the input complex number
	returns a complex number that is the square root of z
*/
__inline__ __device__ hipFloatComplex cuCsqrt(hipFloatComplex z){
	float r = hipCabsf(z);
    float theta = atan2(z.y,z.x);
    hipFloatComplex sqrt_z = make_hipFloatComplex(sqrtf(r) * cosf(theta / 2.0f),
                                                sqrtf(r) * sinf(theta / 2.0f));
	return sqrt_z;
}

/**
	This is the second stage of the matrix inverse.
	It takes the unfinished rows and subtract them with
	the multiplication of the ith column element in the row and the i row. 
	A is the matrix that is choleskylised
	i is the column that is calculated
	N is the nr of rows/columns of the A matrix (NxN)
	The A matrix is overwriten in this function
*/
__global__ void cInv2(float2* A,float2* Ainv, int i, int N){
	//for the column it is N elements.
	int rowElements = N-(i+1);
	int rowthread = blockDim.x * gridDim.x;
	int extraRows = extra(rowElements, rowthread);
	int colthread = blockDim.y * gridDim.y;
	int extraCols = extra(N, colthread);
	
	int row = threadIdx.x + blockDim.x * blockIdx.x; //find what col and row this thread is responsible for
	int col = threadIdx.y + blockDim.y * blockIdx.y;	//ex 0,0 or 1,3
	
	
	for(int v=0;v < extraRows; v++){
		for(int w=0; w<extraCols;w++){
			int tmpRow = row+rowthread*v;
			int tmpCol = col+colthread*w;
			if(tmpRow <= rowElements && tmpCol <=N){
				if(tmpRow+i+1 >= tmpCol){
				Ainv[tmpCol*N+tmpRow+i+1] = hipCsubf(Ainv[tmpCol*N+tmpRow+i+1],hipCmulf(Ainv[tmpCol*N+i],A[i*N+tmpRow+i+1]));
				}
			}
		}
	}
	
    extraRows = extra(N, rowthread);

	int next_i = i+1;
	int index = 0;

	__shared__ float2 sharedDiagonal;
    for(int v=0;v < extraRows; v++){
        int tmpRow = row+rowthread*v;

        int idx = next_i * N + tmpRow + next_i; //find index
        int diagonal = (next_i * N) + next_i; //get diagonal element index

        

        if(idx == diagonal && tmpRow < N-next_i && next_i < N){//part 1, if diagonal element
            sharedDiagonal = cuCsqrt(A[idx]);
			index = idx;
        }
	}
        __syncthreads(); //every thread needs to reach this place before continuing execution
		A[index] = sharedDiagonal;
 for(int v=0;v < extraRows; v++){
        int tmpRow = row+rowthread*v;

        int idx = next_i * N + tmpRow + next_i; //find index
        int diagonal = (next_i * N) + next_i; //get diagonal element index    
    
		if(idx != diagonal && tmpRow < N-next_i && next_i < N){//part 2
            A[idx] = hipCdivf(A[idx], sharedDiagonal);//A[idx]/A[diagonal]
        }
    }
}

/**

	The first stage of column wise matrix inversion.
	In this stage the ith row is devided by its diagonal element
	
   A is the matrix that is choleskylised
   Ainv is the resulting inverted matrix and needs to be an empty matrix
   i is the column that is calculated
   N is the nr of rows/columns of the A matrix (NxN)
*/
__global__ void cInv1(float2* A,float2* Ainv, int i, int N){
	int elements = i+1; 							//elements calculated
	int rowthread = blockDim.x * gridDim.x;			//nr of threads in the row
	int extraRows = extra(elements, rowthread);		//how many elements this thread will run 
	
	int col = threadIdx.x + blockDim.x * blockIdx.x; //find what col and row this thread is responsible for
	for(int v=0;v < extraRows; v++){
		int tmpCol = col+rowthread*v;
		if(tmpCol <= elements){
			if(tmpCol == i){
				Ainv[tmpCol*N+i].x = 1;
			}
			Ainv[tmpCol*N+i] = hipCdivf(Ainv[tmpCol*N+i],A[i*N+i]);
		}
	}
}

/**
   The third step of the block cholesky decomposition where U-c*c^H.
   A is the matrix that is choleskylised
   i is the column that is calculated
   N is the nr of rows/columns of the A matrix (NxN)
   The A matrix is overwriten in this function
*/
__global__ void bChol3(float2* A, float2* Ainv, int i, int N){
	
	int elements = i+1; 							//elements calculated
	int rowthread = blockDim.x * gridDim.x;			//nr of threads in the row
	int extraRows = extra(elements, rowthread);		//how many elements this thread will run 
	
	int col = threadIdx.x + blockDim.x * blockIdx.x; //find what col and row this thread is responsible for
	for(int v=0;v < extraRows; v++){
		int tmpCol = col+rowthread*v;
		if(tmpCol <= elements){
			if(tmpCol == i){
				Ainv[tmpCol*N+i].x = 1;
			}
			Ainv[tmpCol*N+i] = hipCdivf(Ainv[tmpCol*N+i],A[i*N+i]);
		}
	}
	
	//here is inversion pt 2
	elements = N-(i+1);

	extraRows = extra(elements, rowthread);
	int colthread = blockDim.y * gridDim.y;
	int extraCols = extra(elements, colthread);
	
	int j = i+1;
	int row = threadIdx.x + blockDim.x * blockIdx.x; //find what col and row this thread is responsible for
	col = threadIdx.y + blockDim.y * blockIdx.y;
	

	for(int v=0;v < extraRows; v++){
		for(int w=0; w<extraCols;w++){
				int tmpRow = row+rowthread*v;
				int tmpCol = col+colthread*w;
			if(tmpRow >= tmpCol && tmpRow<=elements && tmpCol<=elements){
				float2 tmp = A[(N*i+i+1)+tmpCol];
				tmp.y = -tmp.y;
				A[(tmpCol+j)*N+j+tmpRow] = hipCsubf(A[(tmpCol+j)*N+j+tmpRow],hipCmulf(A[(N*i+i+1)+tmpRow],tmp));
			}
		}
	}
}

/**
   The secons step of the block cholesky decomposition where c=c/d.
   A is the matrix that is choleskylised
   i is the column that is calculated
   N is the nr of rows/columns of the A matrix (NxN)
   elements is the number of elements needed to calculate
   The A matrix is overwriten in this function
*/
__global__ void bChol2(float2* A,const int i,const int N){

    int rowthread = blockDim.x * gridDim.x;
    int extraRows = extra(N, rowthread);

    int row = threadIdx.x + blockDim.x * blockIdx.x; //find what col and row this thread is responsible for

    for(int v=0;v < extraRows; v++){
        int tmpRow = row+rowthread*v;

        int idx = i * N + tmpRow + i; //find index
        int diagonal = (i * N) + i; //get diagonal element index

        __shared__ float2 sharedDiagonal;

        if(idx == diagonal  && tmpRow < N-i){//part 1, if diagonal element
            sharedDiagonal = cuCsqrt(A[idx]);
            A[idx] = sharedDiagonal;
        }
        __syncthreads(); //every thread needs to reach this place before continuing execution
        if(idx != diagonal && tmpRow < N-i){//part 2
            A[idx] = hipCdivf(A[idx], sharedDiagonal);//A[idx]/A[diagonal]
        }
    }
}


/**
   The first and second step of the block cholesky decomposition where sqrt(d) and c=c/d.
   A is the matrix that is choleskylised
   i is the column that is calculated
   N is the nr of rows/columns of the A matrix (NxN)
   The A matrix is overwriten in this function
*/
__global__ void bChol(float2* A,int i,int N){
	A[i*N+i] = cuCsqrt(A[i*N+i]);
}

/**
	This metod takes in a matrix and returns the hermitian transpose of the matrix	
	input_h is the input matrix with size KxN
	output_hh is the resulting matrix with size NxK
	K is the nr of columns in input_h
	N the nr of rows in input_h
*/

__global__ void hermitian_transpose(const float2* input_h, float2* output_hh, int K, int N) { //const because we do not want to modify the input matrix!!!
	
	int rowthread = blockDim.x * gridDim.x;
	int extraRows = extra(N, rowthread);
	int colthread = blockDim.y * gridDim.y;
	int extraCols = extra(K, colthread);
	
	int row = threadIdx.x + blockDim.x * blockIdx.x; //find what col and row this thread is responsible for
	int col = threadIdx.y + blockDim.y * blockIdx.y;	//ex 0,0 or 1,3
	
	for(int v=0;v < extraRows; v++){
		for(int w=0; w<extraCols;w++){
			int tmpRow = row+rowthread*v;
			int tmpCol = col+colthread*w;
			if(tmpRow < N && tmpCol < K){
				
				int idx_in = tmpCol * N + tmpRow;
				int idx_out = tmpRow * K + tmpCol;
				//conjugate here - in a float2: .x is the real part, .y is imaginary part
				output_hh[idx_out].x = input_h[idx_in].x; //conjugate
				output_hh[idx_out].y = -input_h[idx_in].y; //conjugate, it is negative for the imaginary part
			}
		}
	}
}

/**
	Pre Condition: Same size at Arow/Bcol 
	This function calculates the dot product of two complex matrices where A.B=C
	A is the first input matrix
	B is the second input matrix
	C is the resulting matrix
	res_row is the nr of rows in matrix A
	a_col_b_row is the nr of columns of A matrix and nr of rows in B matrix
	res_col is nr of columns in B matrix
*/
__global__ void complex_matrix_mult(const float2* A, const float2* B, float2* C, const int res_row, const int a_col_b_row, const int res_col) {
	int rowthread = blockDim.x * gridDim.x;
	int extraRows = extra(res_row, rowthread);
	int colthread = blockDim.y * gridDim.y;
	int extraCols = extra(res_col, colthread);

	int row = threadIdx.x + blockDim.x * blockIdx.x; 
	int col = threadIdx.y + blockDim.y * blockIdx.y;

	for(int v=0;v < extraRows; v++){
			for(int w=0; w<extraCols;w++){
				int tmpRow = row+rowthread*v;
				int tmpCol = col+colthread*w;
				
				if (tmpRow < res_row && tmpCol < res_col) {		
					float2 sum = make_float2(0.0f, 0.0f);
				
				for (int k = 0; k < a_col_b_row; k++) {
					float2 a = A[k * res_row + tmpRow]; //column-major
					float2 b = B[tmpCol * a_col_b_row + k];
					float real_part = a.x * b.x - a.y * b.y;
					float imag_part = a.x * b.y + a.y * b.x;
					sum.x += real_part;
					sum.y += imag_part;
				}
				C[tmpCol * res_row + tmpRow] = sum;
			}
		}
	}
}

/**
	Pre Condition: Same size at Arow/Bcol 
	This function calculates the dot product of two complex matrices where A.B=C but only the lower tirangle
	A is the first input matrix
	B is the second input matrix
	C is the resulting matrix
	res_row is the nr of rows in matrix A
	a_col_b_row is the nr of columns of A matrix and nr of rows in B matrix
	res_col is nr of columns in B matrix
*/
__global__ void Ltriangle_complex_matrix_mult(const float2* A, const float2* B, float2* C, const int res_row, const int a_col_b_row, const int res_col) {

	int rowthread = blockDim.x * gridDim.x;
	int extraRows = extra(res_row, rowthread);
	int colthread = blockDim.y * gridDim.y;
	int extraCols = extra(res_col, colthread);

	int row = threadIdx.x + blockDim.x * blockIdx.x; 
	int col = threadIdx.y + blockDim.y * blockIdx.y;


	for(int v=0;v < extraRows; v++){
		for(int w=0; w<extraCols;w++){
			int tmpRow = row+rowthread*v;
			int tmpCol = col+colthread*w;
			if (tmpRow < res_row && tmpCol < res_col && tmpRow >= tmpCol) {		
				float2 sum = make_float2(0.0f, 0.0f);
				for (int k = 0; k < a_col_b_row; k++) {
					float2 a = A[k * res_row + tmpRow];
					float2 b = B[tmpCol * a_col_b_row + k];
					float real_part = a.x * b.x - a.y * b.y;
					float imag_part = a.x * b.y + a.y * b.x;
					sum.x += real_part;
					sum.y += imag_part;
				}
				C[tmpCol * res_row + tmpRow] = sum;
			}
		}
	}
}


__global__ void transpose_and_mult(const float2* input_h, float2* output_hh, int K, int N, const float2* A, const float2* B, float2* C, const int res_row, const int a_col_b_row, const int res_col){
	int rowthread = blockDim.x * gridDim.x;
	int extraRows = extra(res_row, rowthread);
	int colthread = blockDim.y * gridDim.y;
	int extraCols = extra(res_col, colthread);

	int row = threadIdx.x + blockDim.x * blockIdx.x; 
	int col = threadIdx.y + blockDim.y * blockIdx.y;

	for(int v=0;v < extraRows; v++){
			for(int w=0; w<extraCols;w++){
				int tmpRow = row+rowthread*v;
				int tmpCol = col+colthread*w;
				
				if (tmpRow < res_row && tmpCol < res_col) {		
					float2 sum = make_float2(0.0f, 0.0f);
				
				for (int k = 0; k < a_col_b_row; k++) {
					float2 a = A[k * res_row + tmpRow]; //column-major
					float2 b = B[tmpCol * a_col_b_row + k];
					float real_part = a.x * b.x - a.y * b.y;
					float imag_part = a.x * b.y + a.y * b.x;
					sum.x += real_part;
					sum.y += imag_part;
				}
				C[tmpCol * res_row + tmpRow] = sum;
			}
		}
	}
	
	extraRows = extra(N, rowthread);
	extraCols = extra(K, colthread);
	
	for(int v=0;v < extraRows; v++){
		for(int w=0; w<extraCols;w++){
			int tmpRow = row+rowthread*v;
			int tmpCol = col+colthread*w;
			if(tmpRow < N && tmpCol < K){
				
				int idx_in = tmpCol * N + tmpRow;
				int idx_out = tmpRow * K + tmpCol;
				//conjugate here - in a float2: .x is the real part, .y is imaginary part
				output_hh[idx_out].x = input_h[idx_in].x; //conjugate
				output_hh[idx_out].y = -input_h[idx_in].y; //conjugate, it is negative for the imaginary part
			}
		}
	}
	
}

int main() {
	//read the Y.csv
	//128x8
	int K,N,blockSize,gridSize;
	//int K=128,N=4096,blockSize=32,gridSize=8;
	int nrOfFrames;
	
	printf("Enter N K blockSize gridSize nrOfFrames\n");
    scanf("%d %d %d %d %d",&N,&K,&blockSize,&gridSize,&nrOfFrames);
	//scanf("%d",&nrOfFrames);
	//int amount = 16;
	//int frameSet[amount] = {1,2,3,4,5,7,10,20,30,40,50,60,70,80,90,100,};
//for(int setOfFrames = 0; setOfFrames<amount;setOfFrames++){
	//nrOfFrames = frameSet[setOfFrames];
	
	printf("Info: %dx%d, blockSize=%d, gridSize=%d, nrOfFrames=%d\n",N,K,blockSize,gridSize,nrOfFrames);
	float elapsed_time_ms[10];       		// which is applicable for asynchronous code also
for(int loop=0;loop<10;loop++){	
	// read csv files
	char file1[32] = "";
	sprintf(file1, "%dx%d/Y", N,K);
	float2 *hY;
	hY = read_matrix_from_csv(file1, N, 1);
	
	float2 *H;
	sprintf(file1, "%dx%d/H", N,K);
	H = read_matrix_from_csv(file1, N, K);
	
	hipStream_t *streams = (hipStream_t *) malloc(nrOfFrames * sizeof(hipStream_t));
	for(int frame = 0; frame < nrOfFrames; frame++){
		hipStreamCreate(&streams[frame]);
	}

	
	hipEvent_t start, stop;     		// using cuda events to measure time
	//float elapsed_time_ms;       		// which is applicable for asynchronous code also
	
	hipEventCreate(&start);     		// instrument code to measure start time
	hipEventCreate(&stop);
		
	//The h stands for host
	float2 *hHHY[nrOfFrames];


	for(int frame = 0; frame<nrOfFrames; frame++){
		hHHY[frame] = (float2 *) malloc(K * sizeof(float2));
	}
	
	//cudaEventRecord(start, 0);//start time +memcpy overhead
	//The d stands for device
	
	hipFloatComplex *dH[nrOfFrames], *dHH[nrOfFrames], *dmHH[nrOfFrames], *dInv[nrOfFrames], *dInvH[nrOfFrames],*dInvM[nrOfFrames],*dY[nrOfFrames],*dHHY[nrOfFrames],*dx[nrOfFrames];
	for(int frame = 0; frame<nrOfFrames; frame++){

		hipMalloc((void **)&dH[frame], N*K*sizeof(hipFloatComplex));
		hipMalloc((void **)&dHH[frame],  K*N*sizeof(hipFloatComplex));
		hipMalloc((void **)&dmHH[frame], K*K*sizeof(hipFloatComplex));
		hipMalloc((void **)&dInv[frame], K*K*sizeof(hipFloatComplex));
		hipMalloc((void **)&dInvH[frame], K*K*sizeof(hipFloatComplex));
		hipMalloc((void **)&dInvM[frame], K*K*sizeof(hipFloatComplex));
		hipMalloc((void **)&dY[frame], N*sizeof(hipFloatComplex));
		hipMalloc((void **)&dHHY[frame], K*sizeof(hipFloatComplex));
		hipMalloc((void **)&dx[frame], K*sizeof(hipFloatComplex));
	}	
		
	for(int frame=0; frame<nrOfFrames;frame++){
		//Copy input data to array on GPU.
		hipMemcpyAsync(dH[frame], H, K*N*sizeof(hipFloatComplex), hipMemcpyHostToDevice,streams[frame]);
		hipMemcpyAsync(dY[frame], hY, N*sizeof(hipFloatComplex), hipMemcpyHostToDevice,streams[frame]);
		
	}
	
	hipEventRecord(start, 0);//start time
	//Run the transpose on gpu
	//Number of threads are N*K with N-rows and K-columns
	dim3 blockDims(blockSize,blockSize);
	dim3 GridDims(gridSize,gridSize);
	
	#pragma omp parallel for
	for(int frame = 0; frame<nrOfFrames; frame++){
		hermitian_transpose<<<GridDims,blockDims,0,streams[frame]>>>(dH[frame], dHH[frame],K,N);
	}

	#pragma omp parallel for
	for(int frame = 0; frame<nrOfFrames; frame++){
	//Number of threads are K*K
		Ltriangle_complex_matrix_mult<<<GridDims,blockDims,0,streams[frame]>>>(dHH[frame], dH[frame], dmHH[frame],K,N,K);	
	}
	
	int Block_Dim_x = K;
	int Block_Dim_y = 1;
	dim3 Block2(Block_Dim_x,Block_Dim_y);
	
	for(int i = 0; i < K; i++){	
		#pragma omp parallel for
		for(int frame = 0; frame<nrOfFrames; frame++){
			if(i==0){
				bChol2<<<1,Block2,sizeof(float2)*(1), streams[frame]>>>(dmHH[frame],i,K); //diagonal is in shared mem
			}
			bChol3<<<1,Block2, 0, streams[frame]>>>(dmHH[frame],dInv[frame],i,K);
			cInv2<<<1,Block2, sizeof(float2)*(1), streams[frame]>>>(dmHH[frame],dInv[frame],i,K);
		}

	}

	//#pragma unroll 10
	#pragma omp parallel for
	for(int frame = 0; frame<nrOfFrames; frame++){
		//This part takes the inv of L multiplied with itsef to become A^-1
		transpose_and_mult<<<GridDims,blockDims,0,streams[frame]>>>(dInv[frame], dInvH[frame],K,K, dHH[frame], dY[frame], dHHY[frame],K,N,1);
	}
	//cudaDeviceSynchronize();
	//#pragma unroll 10
	#pragma omp parallel for
	for(int frame = 0; frame<nrOfFrames; frame++){
		complex_matrix_mult<<<GridDims,blockDims,0,streams[frame]>>>(dInvH[frame], dInv[frame], dInvM[frame],K,K,K);
	}

	#pragma omp parallel for
	for(int frame = 0; frame<nrOfFrames; frame++){
		//dHH = 8x8 dHHY = 8x1
		complex_matrix_mult<<<GridDims,blockDims,0,streams[frame]>>>(dInvM[frame], dHHY[frame], dx[frame],K,K,1);		
	}
	
	hipEventRecord(stop, 0);     	// instrument code to measue end time
	hipEventSynchronize(stop);
	for(int frame = 0; frame<nrOfFrames; frame++){
		hipMemcpyAsync(hHHY[frame], dx[frame], K*sizeof(hipFloatComplex), hipMemcpyDeviceToHost,streams[frame]);
	}
	hipDeviceSynchronize();
	//cudaEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms[loop], start, stop);

	for(int i =0; i<nrOfFrames; i++){
		hipStreamSynchronize(streams[i]);
		
	}

	//PRINT RESULT_________________________________________________________
	if(loop==0 /*&& setOfFrames==1*/){
		printf("result x: -------------------------------------------------------------------------\n");
	float2 temp[K];
	//for (int frame = 0; frame<nrOfFrames; frame++){
		//printf("%d \n", i);
		memcpy(temp,hHHY[0],K*sizeof(hipFloatComplex));
		//temp = *hHHY[frame]; 
		for(int i = 0; i<K; ++i) {
			printf("%f+%fi \n", temp[i].x,temp[i].y);//(*hHHY[i*N + frame]).x,(*hHHY[i*N+frame]).y);
		}
	//}
	}
	
	
	
	    // Clean up CUDA streams
    for (int i = 0; i < nrOfFrames; ++i) {
        hipStreamDestroy(streams[i]);
    }
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	
	// Free up the arrays on the GPU.
	hipFree(dH);
	hipFree(dHH);
	hipFree(dmHH);
	hipFree(dInv);
	hipFree(dInvH);
	hipFree(dInvM);
	hipFree(dY);
	hipFree(dHHY);
	
	//Free from CPU
	free(hY);
	free(H);
}
for(int i=0;i<10;i++)
		printf("Time to calculate results on GPU: %f ms or %f each.\n", elapsed_time_ms[i],elapsed_time_ms[i]/nrOfFrames);
//}
	return 0;
}
