
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_complex.h>

/**
	This is to use qsort
*/
int cmpfunc (const void * a, const void * b) {
   return ( *(int*)a - *(int*)b );
}

/**
	Takes a csv containing a matrix and returns an array column major
*/
float2 *read_matrix_from_csv(char filename[], int num_rows, int num_cols) {
	// Allocate memory for the matrix
	float2 *matrix = (float2 *) malloc(num_rows * num_cols * sizeof(float2));
	bool real = true;

	char tempchar[20];
	strcpy(tempchar, filename);
	for(int i=0;i<2;i++){
		if(i == 0){
			strcat(filename, "_real.csv");
			real = true;
		}
		else{
			strcat(tempchar, "_imag.csv");
			strcpy(filename, tempchar);
			real = false;
		}
		
	  FILE *file = fopen(filename, "r");
	  if (file == NULL) {
		fprintf(stderr, "Error: Could not open file '%s'\n", filename);
		exit(1);
	  }
	  // Read the data from the file into the matrix
	  int size = (num_rows*20+1)*sizeof(char);//max 20 chars and "," for one value + "\0" or "\n"
	  char line[size];
	  int row = 0, col = 0;
	  while (fgets(line, size, file) && row < num_rows) {
		if (line[strlen(line) - 1] == '\n') {
		  line[strlen(line) - 1] = '\0';  // Remove newline character
		}

		char *token = strtok(line, ",");
		while (token != NULL && col < num_cols) {
			if(real){
				matrix[row + col * num_rows].x = atof(token); // Change the ordering of the matrix
			}
			else{
				matrix[row + col * num_rows].y = atof(token); // Change the ordering of the matrix
			}
			col++;
			token = strtok(NULL, ",");
		}
		col = 0;
		row++;
	  }
	  fclose(file);
	}
  
  return matrix;
}

/**
	extra calculates how many elements of a matrix each thread needs to calculate of there are too few threads
	elements is the number of elements there is in a row/column in the matrix.
	nrOfThreads are the number of threads available for use
*/
__device__ int extra(int elements, int nrOfThreads){
	int tmp = ceil((float)elements/(float)nrOfThreads);
	return tmp;
}

/**
	cuCsqrt takes in a complex number and returns the square root of this number
	z the input complex number
	returns a complex number that is the square root of z
*/
__device__ hipFloatComplex cuCsqrt(hipFloatComplex z){
	float r = hipCabsf(z);
    float theta = atan2(z.y,z.x);
    hipFloatComplex sqrt_z = make_hipFloatComplex(sqrtf(r) * cosf(theta / 2.0f),
                                                sqrtf(r) * sinf(theta / 2.0f));
	return sqrt_z;
}

/**
	This is the second stage of the matrix inverse.
	It takes the unfinished rows and subtract them with
	the multiplication of the ith column element in the row and the i row. 
	A is the matrix that is choleskylised
	i is the column that is calculated
	N is the nr of rows/columns of the A matrix (NxN)
	The A matrix is overwriten in this function
*/
__global__ void cInv2(float2* A,float2* Ainv, int i, int N){
	//for the column it is N elements.
	int rowElements = N-(i+1);
	int rowthread = blockDim.x * gridDim.x;
	int extraRows = extra(rowElements, rowthread);
	int colthread = blockDim.y * gridDim.y;
	int extraCols = extra(N, colthread);
	
	int row = threadIdx.x + blockDim.x * blockIdx.x; //find what col and row this thread is responsible for
	int col = threadIdx.y + blockDim.y * blockIdx.y;	//ex 0,0 or 1,3
	for(int v=0;v < extraRows; v++){
		for(int w=0; w<extraCols;w++){
			int tmpRow = row+rowthread*v;
			int tmpCol = col+colthread*w;
			if(tmpRow <= rowElements && tmpCol <=N){
				if(tmpRow+i+1 >= tmpCol){
				Ainv[tmpCol*N+tmpRow+i+1] = hipCsubf(Ainv[tmpCol*N+tmpRow+i+1],hipCmulf(Ainv[tmpCol*N+i],A[i*N+tmpRow+i+1]));
				}
			}
		}
	}	
}

/**

	The first stage of column wise matrix inversion.
	In this stage the ith row is devided by its diagonal element
	
   A is the matrix that is choleskylised
   Ainv is the resulting inverted matrix and needs to be an empty matrix
   i is the column that is calculated
   N is the nr of rows/columns of the A matrix (NxN)
*/
__global__ void cInv1(float2* A,float2* Ainv, int i, int N){
	int elements = i+1; 							//elements calculated
	int rowthread = blockDim.x * gridDim.x;			//nr of threads in the row
	int extraRows = extra(elements, rowthread);		//how many elements this thread will run 
	
	int col = threadIdx.x + blockDim.x * blockIdx.x; //find what col and row this thread is responsible for
	for(int v=0;v < extraRows; v++){
		int tmpCol = col+rowthread*v;
		if(tmpCol <= elements){
			if(tmpCol == i){
				Ainv[tmpCol*N+i].x = 1;
			}
			Ainv[tmpCol*N+i] = hipCdivf(Ainv[tmpCol*N+i],A[i*N+i]);
		}
	}
}

/**
   The third step of the block cholesky decomposition where U-c*c^H.
   A is the matrix that is choleskylised
   i is the column that is calculated
   N is the nr of rows/columns of the A matrix (NxN)
   The A matrix is overwriten in this function
*/
__global__ void bChol3(float2* A, int i, int N){
	
	//N-(i+1) is the number of elements run in both x and y
	int elements = N-(i+1);
	int rowthread = blockDim.x * gridDim.x;
	int extraRows = extra(elements, rowthread);
	int colthread = blockDim.y * gridDim.y;
	int extraCols = extra(elements, colthread);
	
	int j = i+1;
	int row = threadIdx.x + blockDim.x * blockIdx.x; //find what col and row this thread is responsible for
	int col = threadIdx.y + blockDim.y * blockIdx.y;
	for(int v=0;v < extraRows; v++){
		for(int w=0; w<extraCols;w++){
				int tmpRow = row+rowthread*v;
				int tmpCol = col+colthread*w;
			if(tmpRow >= tmpCol && tmpRow<=elements && tmpCol<=elements){
				float2 tmp = A[(N*i+i+1)+tmpCol];
				tmp.y = -tmp.y;
				A[(tmpCol+j)*N+j+tmpRow] = hipCsubf(A[(tmpCol+j)*N+j+tmpRow],hipCmulf(A[(N*i+i+1)+tmpRow],tmp));
			}
		}
	}	
}

/**
   The secons step of the block cholesky decomposition where c=c/d.
   A is the matrix that is choleskylised
   i is the column that is calculated
   N is the nr of rows/columns of the A matrix (NxN)
   elements is the number of elements needed to calculate
   The A matrix is overwriten in this function
*/
__global__ void bChol2(float2* A,int i,int N){
	
	int rowthread = blockDim.x * gridDim.x;
	//N-(i+1) is the number of elements calculated in this part
	int elements = N-(i+1);
	int extraRows = extra(elements, rowthread);
	
	int row = blockIdx.x + 1;
	
	for(int v=0;v < extraRows; v++){
		int tmpRow = row+rowthread*v;
		if(tmpRow <= elements){
			A[(i*N+i)+tmpRow] = hipCdivf(A[(i*N+i)+tmpRow], A[i*N+i]);
		}
	}
}

/**
   The first and second step of the block cholesky decomposition where sqrt(d) and c=c/d.
   A is the matrix that is choleskylised
   i is the column that is calculated
   N is the nr of rows/columns of the A matrix (NxN)
   The A matrix is overwriten in this function
*/
__global__ void bChol(float2* A,int i,int N){
	A[i*N+i] = cuCsqrt(A[i*N+i]);
}

/**
	This metod takes in a matrix and returns the hermitian transpose of the matrix	
	input_h is the input matrix with size KxN
	output_hh is the resulting matrix with size NxK
	K is the nr of columns in input_h
	N the nr of rows in input_h
*/

__global__ void hermitian_transpose(const float2* input_h, float2* output_hh, int K, int N) { //const because we do not want to modify the input matrix!!!
	
	int rowthread = blockDim.x * gridDim.x;
	int extraRows = extra(N, rowthread);
	int colthread = blockDim.y * gridDim.y;
	int extraCols = extra(K, colthread);
	
	int row = threadIdx.x + blockDim.x * blockIdx.x; //find what col and row this thread is responsible for
	int col = threadIdx.y + blockDim.y * blockIdx.y;	//ex 0,0 or 1,3
	
	for(int v=0;v < extraRows; v++){
		for(int w=0; w<extraCols;w++){
			int tmpRow = row+rowthread*v;
			int tmpCol = col+colthread*w;
			if(tmpRow < N && tmpCol < K){
				
				int idx_in = tmpCol * N + tmpRow;
				int idx_out = tmpRow * K + tmpCol;
				//printf("(%d,%d)  in: %d, out: %d\n",tmpRow,tmpCol,idx_in,idx_out);
				//conjugate here - in a float2: .x is the real part, .y is imaginary part
				output_hh[idx_out].x = input_h[idx_in].x; //conjugate
				output_hh[idx_out].y = -input_h[idx_in].y; //conjugate, it is negative for the imaginary part
			}
		}
	}
}

/**
	Pre Condition: Same size at Arow/Bcol 
	This function calculates the dot product of two complex matrices where A.B=C
	A is the first input matrix
	B is the second input matrix
	C is the resulting matrix
	res_row is the nr of rows in matrix A
	a_col_b_row is the nr of columns of A matrix and nr of rows in B matrix
	res_col is nr of columns in B matrix
*/
__global__ void complex_matrix_mult(const float2* A, const float2* B, float2* C, const int res_row, const int a_col_b_row, const int res_col) {
	int rowthread = blockDim.x * gridDim.x;
	int extraRows = extra(res_row, rowthread);
	int colthread = blockDim.y * gridDim.y;
	int extraCols = extra(res_col, colthread);

	int row = threadIdx.x + blockDim.x * blockIdx.x; 
	int col = threadIdx.y + blockDim.y * blockIdx.y;

	for(int v=0;v < extraRows; v++){
			for(int w=0; w<extraCols;w++){
				int tmpRow = row+rowthread*v;
				int tmpCol = col+colthread*w;
				
				if (tmpRow < res_row && tmpCol < res_col) {		
					float2 sum = make_float2(0.0f, 0.0f);
				
				for (int k = 0; k < a_col_b_row; k++) {
					float2 a = A[k * res_row + tmpRow]; //column-major
					float2 b = B[tmpCol * a_col_b_row + k];
					float real_part = a.x * b.x - a.y * b.y;
					float imag_part = a.x * b.y + a.y * b.x;
					sum.x += real_part;
					sum.y += imag_part;
				}
				C[tmpCol * res_row + tmpRow] = sum;
			}
		}
	}
}

/**
	Pre Condition: Same size at Arow/Bcol 
	This function calculates the dot product of two complex matrices where A.B=C but only the lower tirangle
	A is the first input matrix
	B is the second input matrix
	C is the resulting matrix
	res_row is the nr of rows in matrix A
	a_col_b_row is the nr of columns of A matrix and nr of rows in B matrix
	res_col is nr of columns in B matrix
*/
__global__ void Ltriangle_complex_matrix_mult(const float2* A, const float2* B, float2* C, const int res_row, const int a_col_b_row, const int res_col) {

	int rowthread = blockDim.x * gridDim.x;
	int extraRows = extra(res_row, rowthread);
	int colthread = blockDim.y * gridDim.y;
	int extraCols = extra(res_col, colthread);

	int row = threadIdx.x + blockDim.x * blockIdx.x; 
	int col = threadIdx.y + blockDim.y * blockIdx.y;


	for(int v=0;v < extraRows; v++){
		for(int w=0; w<extraCols;w++){
			int tmpRow = row+rowthread*v;
			int tmpCol = col+colthread*w;
			if (tmpRow < res_row && tmpCol < res_col && tmpRow >= tmpCol) {		
				float2 sum = make_float2(0.0f, 0.0f);
				for (int k = 0; k < a_col_b_row; k++) {
					float2 a = A[k * res_row + tmpRow];
					float2 b = B[tmpCol * a_col_b_row + k];
					float real_part = a.x * b.x - a.y * b.y;
					float imag_part = a.x * b.y + a.y * b.x;
					sum.x += real_part;
					sum.y += imag_part;
				}
				C[tmpCol * res_row + tmpRow] = sum;
			}
		}
	}
}

int main() {
	//read the Y.csv
	//128x8
	int K,N,blockSize,gridSize;
	//int K=1024,N=128,blockSize=32,gridSize=4;
	int nrOfFrames;
	
	printf("Enter N K blockSize gridSize nrOfFrames\n");
    scanf("%d %d %d %d %d",&N,&K,&blockSize,&gridSize,&nrOfFrames);
	//scanf("%d",&nrOfFrames);
	
	printf("Info: %dx%d, blockSize=%d, gridSize=%d, nrOfFrames=%d\n",N,K,blockSize,gridSize,nrOfFrames);
	
	// read csv files
	char file1[32] = "";
	sprintf(file1, "%dx%d/Y", N,K);
	float2 *hY;
	hY = read_matrix_from_csv(file1, N, 1);
	
	float2 *H;
	sprintf(file1, "%dx%d/H", N,K);
	H = read_matrix_from_csv(file1, N, K);
	
	
	hipEvent_t start, stop;     		// using cuda events to measure time
	float elapsed_time_ms;       		// which is applicable for asynchronous code also
	
	hipEventCreate(&start);     		// instrument code to measure start time
	hipEventCreate(&stop);
		
	//The h stands for host
	float2 *hHHY[nrOfFrames];
	
	for(int frame = 0; frame<nrOfFrames; frame++){
		hHHY[frame] = (float2 *) malloc(K * sizeof(float2));
	}
	
	hipEventRecord(start, 0);//start time
	//The d stands for device
	hipFloatComplex *dH[nrOfFrames], *dHH[nrOfFrames], *dmHH[nrOfFrames], *dInv[nrOfFrames], *dInvH[nrOfFrames],*dInvM[nrOfFrames],*dY[nrOfFrames],*dHHY[nrOfFrames],*dx[nrOfFrames];
	for(int frame = 0; frame<nrOfFrames; frame++){
		hipMalloc((void **)&dH[frame], N*K*sizeof(hipFloatComplex));
		hipMalloc((void **)&dHH[frame],  K*N*sizeof(hipFloatComplex));
		hipMalloc((void **)&dmHH[frame], K*K*sizeof(hipFloatComplex));
		hipMalloc((void **)&dInv[frame], K*K*sizeof(hipFloatComplex));
		hipMalloc((void **)&dInvH[frame], K*K*sizeof(hipFloatComplex));
		hipMalloc((void **)&dInvM[frame], K*K*sizeof(hipFloatComplex));
		hipMalloc((void **)&dY[frame], N*sizeof(hipFloatComplex));
		hipMalloc((void **)&dHHY[frame], K*sizeof(hipFloatComplex));
		hipMalloc((void **)&dx[frame], K*sizeof(hipFloatComplex));
	}	
		
	for(int frame=0; frame<nrOfFrames;frame++){
		//Copy input data to array on GPU.
		hipMemcpy(dH[frame], H, K*N*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
		hipMemcpy(dY[frame], hY, N*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
		
	}

	//Run the transpose on gpu
	//Number of threads are N*K with N-rows and K-columns
	dim3 blockDims(blockSize,blockSize);
	dim3 GridDims(gridSize,gridSize);
	
	for(int frame = 0; frame<nrOfFrames; frame++){
		hermitian_transpose<<<blockDims,GridDims>>>(dH[frame], dHH[frame],K,N);
	}

	float2 *resultHH;
	resultHH = (float2 *) malloc(K*N * sizeof(float2));
	hipMemcpy(resultHH,dH[0],K*N*sizeof(float2),hipMemcpyDeviceToHost);
	
	
	for(int frame = 0; frame<nrOfFrames; frame++){
	//Number of threads are K*K
		Ltriangle_complex_matrix_mult<<<blockDims,GridDims>>>(dHH[frame], dH[frame], dmHH[frame],K,N,K);	
	}
	hipDeviceSynchronize();
	
	float2 *resultHHH;
	resultHHH = (float2 *) malloc(K*K * sizeof(float2));
	hipMemcpy(resultHHH,dmHH[0],K*K*sizeof(float2),hipMemcpyDeviceToHost);
	
	
	for(int i = 0; i < K; i++){
		//part1 of cholesky. (Diagonal element) one thread
		for(int frame = 0; frame<nrOfFrames; frame++){
			bChol<<<1,1>>>(dmHH[frame],i,K);
		}
		hipDeviceSynchronize();
		//Part2 of cholesky (column compleeted)
		//the amount of threads is getting smaller each iteration
		//it is the number of elements in the vector under the diagonal element
		for(int frame = 0; frame<nrOfFrames; frame++){
			bChol2<<<blockSize,1>>>(dmHH[frame],i,K);
		}
		hipDeviceSynchronize();
		//Part3 of cholesky and start cInv part1
		for(int frame = 0; frame<nrOfFrames; frame++){
			cInv1<<<blockSize,1>>>(dmHH[frame],dInv[frame],i,K);
			bChol3<<<blockDims,GridDims>>>(dmHH[frame],i,K);
		}
		hipDeviceSynchronize();
		//Part2 of inv
		for(int frame = 0; frame<nrOfFrames; frame++){
			cInv2<<<blockDims,GridDims>>>(dmHH[frame],dInv[frame],i,K);
		}
	}	
	
	for(int frame = 0; frame<nrOfFrames; frame++){
		//This part takes the inv of L multiplied with itsef to become A^-1
		hermitian_transpose<<<blockDims,GridDims>>>(dInv[frame], dInvH[frame],K,K);
	}
	hipDeviceSynchronize();
	for(int frame = 0; frame<nrOfFrames; frame++){
		complex_matrix_mult<<<blockDims,GridDims>>>(dInvH[frame], dInv[frame], dInvM[frame],K,K,K);
	
		//dHH = 8x128 dy = 128x1 dHHY = 8x1
		complex_matrix_mult<<<blockDims,GridDims>>>(dHH[frame], dY[frame], dHHY[frame],K,N,1);
	}
	hipDeviceSynchronize();
	for(int frame = 0; frame<nrOfFrames; frame++){
		//dHH = 8x8 dHHY = 8x1
		complex_matrix_mult<<<blockDims,GridDims>>>(dInvM[frame], dHHY[frame], dx[frame],K,K,1);		
	}
	for(int frame = 0; frame<nrOfFrames; frame++){
		hipMemcpy(hHHY[frame], dx[frame], K*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
	}
	
	hipEventRecord(stop, 0);     	// instrument code to measue end time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop);
	
	//PRINT RESULT_________________________________________________________
	
/*		printf("result x: -------------------------------------------------------------------------\n");
	float2 temp[K];
	for (int frame = 0; frame<1; frame++){
		//printf("%d \n", i);
		memcpy(temp,hHHY[frame],K*sizeof(cuFloatComplex));
		//temp = *hHHY[frame]; 
		for(int i = 0; i<K; ++i) {
			printf("%f+%fi \n", temp[i].x,temp[i].y);//(*hHHY[i*N + frame]).x,(*hHHY[i*N+frame]).y);
		}
	}
	*/
	printf("Time to calculate results on GPU: %f ms or %f each.\n", elapsed_time_ms,elapsed_time_ms/nrOfFrames);
	
	// Free up the arrays on the GPU.
	hipFree(dH);
	hipFree(dHH);
	hipFree(dmHH);
	hipFree(dInv);
	hipFree(dInvH);
	hipFree(dInvM);
	hipFree(dY);
	hipFree(dHHY);
	
	//Free from CPU
	free(hY);
	free(H);

    return 0;
}
